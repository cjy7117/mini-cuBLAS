
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <time.h>
#include <stdio.h>
#include <cmath>
#define TEST_RUN 100
#define ESP 10e-10
using namespace std;


__global__ void
dgemm_kernel2(int m, int n, int k, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel2_sass(int m, int n, int k,
	      double * A, int lda,
	      double * B, int ldb,
	      double * C, int ldc);

__global__ void
dgemm_kernel2_1(int m, int n, int k, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

__global__ void
dgemm_kernel3(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, int t,
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

__global__ void
dgemm_kernel4_3(int m, int n, int k, int T, int t, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

void check_C(double * dC, int m, double * checkC);

void test_cublas_mv(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_cublas_mm(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel2_sass(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4_1(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4_2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4_3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test(int m, int k);

int main(){
	for (int i = 128; i <= 32768; i *= 2){
		//i = 20480;
		cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 1 << ")" << endl;
		test(i, i);
	}
}

void test(int m, int k){
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 1;
    //int k = 20480;
    double * A = new double[m * k];
    double * B = new double[n * k];
    double * C = new double[m * n];  
    double * checkC = new double[m * n];   

    for (int i = 0;i < m * k; i++){
    	A[i] = i;
    }

    //    for (int i = 0; i < m; i++){
    // for (int j = 0; j < k; j++){
    //	cout << *( A + i + j * m) << " ";
    // }
    // cout << endl;
    //}
    
    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    double * dA;
    hipMalloc(&dA, m * k * sizeof(double));
    int lda = m;

    double * dB; 
    hipMalloc(&dB,  n * k * sizeof(double));
    int ldb = k;

    double * dC;
    hipMalloc(&dC, m * n * sizeof(double));
    int ldc = m;

    double * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(double));

    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    
   

    test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
    test_cublas_mv(m, n, k, dA, lda, dB, ldb,  dC, ldc);
    //test_kernel2(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel2_sass(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel2_1(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel3(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel4(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel4_1(m, n, k, dA, lda, dB, ldb, dC, ldc);
    //test_kernel4_2(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel4_3(m, n, k, dA, lda, dB, ldb, dC, ldc);
    


   
    hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(checkC, dcheckC, m * n * sizeof(double), hipMemcpyDeviceToHost);

    check_C(C, m, checkC);    
    
    //for (int i = 0; i < m * n; i++){
    // cout<<C[i]<<" ";	
    //}
    
    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}


void test_cublas_mv(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);


    int incb = 1;
    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemv(handle, HIPBLAS_OP_N, m, k,
      			  &one, dA, lda, dB, incb, &zero, dC, incb);

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of culasdgemv:" << real_time <<" ms." << endl;

}


void test_cublas_mm(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){

    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

 
    int incb = 1;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
	  	  &one, dA, lda, dB, ldb, &zero, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of culasdgemm:" << real_time <<" ms." << endl;
}

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){


    int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();

    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, 
							dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;
    
    
    float mem_access = (2*m*n*sizeof(double)*TEST_RUN)/1e9;
   
    cout <<"Runing time of dgemm_kernel2: " << real_time << " ms." <<endl;    
    
} 

void test_kernel2_sass(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc){


    int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();

    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2_sass<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, 
              dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2_sass: " << real_time << " ms." << endl;    

} 


void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){
	


	int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock(); 
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2_1<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
    						  dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2_1: " << real_time << " ms." << endl;


}

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel3<<<blocksPerGrid, threadsPerBlock,  T * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipDeviceSynchronize();
    t = clock() - t;

    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel3: " << real_time << " ms." << endl;	    
}


void test_kernel4(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4<<<blocksPerGrid, threadsPerBlock, ((T) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    }
    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;    
}

void test_kernel4_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_1<<<blocksPerGrid, threadsPerBlock, ((T) + (T * tt)) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4_1: " << real_time << " ms." << endl;	  

}


void test_kernel4_2(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_2<<<blocksPerGrid, threadsPerBlock, (T * tt) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4_2: " << real_time << " ms." << endl;	  

}


void test_kernel4_3(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_3<<<blocksPerGrid, threadsPerBlock, ((T) + (T * tt)) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4_3: " << real_time << " ms." << endl;	  

}

void check_C(double * dC, int m, double * checkC) {
	for (int i = 0; i < m; i++){
		if (fabs(dC[i] - checkC[i]) > ESP){
			cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
			return;
		}
	}
	cout << "correct" << endl;
}


__global__ void
dgemm_kernel2(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{

        
	//determine the row to process
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	A = A + idx;
	double a;
	double b;
	double temp = 0;
	for (int i = 0;i < k; i++){
	  a = *(A + i * lda);
	  b = *(B + i);
	  temp = temp + a * b;
	}
	 *(C + idx) = temp;
	
}

__global__ void
dgemm_kernel2_sass(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                                                                   
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register double a;
  register double b;
  register double * AA =A;
  register double * BB = B;
  register double temp = 0;
  register long long int lda1 = lda * 8;
  for (int i = 0;i < k; i++){
    asm volatile ("{\n\t"
		  "ld.global.f64 %0, [%2];\n\t"
		  "ld.global.f64 %1, [%3];\n\t"
		  "fma.rz.f64 %4,%0,%1,%4;\n\t"
		  "add.u64 %2, %2, %5;\n\t"
		  "add.u64 %3, %3, 0x8;\n\t"
		  "}"
		  : "+d"(a), "+d"(b), "+l"(AA), "+l"(BB),
		    "+d"(temp):"l"(lda1)
		 );
  }
  *(C + idx) = temp;

}


__global__ void
dgemm_kernel2_1(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp = 0;
  double a = 0;
  double b = 0;
  for (int i = 0; i < k; i++){
    a = *(A + i * lda);
    b = *(B + i);
    temp += a * b;
  }
  *(C + idx) = temp;
}

__global__ void
dgemm_kernel3(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  
  double a = 0;
  for (int j = 0; j < k; j += T){
    cache[threadIdx.x] = *(B + threadIdx.x);
    B += T;
    __syncthreads();
    for (int i = 0; i < T; i++) {
      a = *(A + (i + j) * lda);
      temp1 += a * cache[i];
    }
    __syncthreads();

  }
  *(C + idx) = temp1;
}

__global__ void
dgemm_kernel4(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
                                                                                                                                                                                      
  //prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
 
  double r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13, r14, r15;

  for (int j = 0; j < k; j += T){
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    __syncthreads();
    B += T;
	
	if (j + T < k) {  
	    A = A + T * lda;
	    
	    r0 = *(A + 0 *lda);
	    r1 = *(A + 1 *lda);
	    r2 = *(A + 2 *lda);
	    r3 = *(A + 3 *lda);   
	    r4 = *(A + 4 *lda);
	    r5 = *(A+ 5 *lda);
	    r6 = *(A + 6 *lda);
	    r7 = *(A + 7 *lda);

	    r8 = *(A + 8 *lda);
	    r9 = *(A + 9 *lda);
	    r10 = *(A + 10 *lda);
	    r11 = *(A + 11 *lda);
	    r12 = *(A + 12 *lda);
	    r13 = *(A + 13 *lda);
	    r14 = *(A + 14 *lda);
	    r15 = *(A + 15 *lda);
	}
	 
    for (int i = 0; i < T; i++) {
      temp1 += cacheA[threadIdx.x + i * T] * cacheB[i];
    }

    if (j + T < k) {
	    cacheA[threadIdx.x + 0 * T] = r0;
	    cacheA[threadIdx.x + 1 * T] = r1;
	    cacheA[threadIdx.x + 2 * T] = r2;
	    cacheA[threadIdx.x + 3 * T] = r3;
	    cacheA[threadIdx.x + 4 * T] = r4;
	    cacheA[threadIdx.x + 5 * T] = r5;
	    cacheA[threadIdx.x + 6 * T] = r6;
	    cacheA[threadIdx.x + 7 * T] = r7;

	    cacheA[threadIdx.x + 8 * T] = r8;
	    cacheA[threadIdx.x + 9 * T] = r9;
	    cacheA[threadIdx.x + 10 * T] = r10;
	    cacheA[threadIdx.x + 11 * T] = r11;
	    cacheA[threadIdx.x + 12 * T] = r12;
	    cacheA[threadIdx.x + 13 * T] = r13;
	    cacheA[threadIdx.x + 14 * T] = r14;
	    cacheA[threadIdx.x + 15 * T] = r15;
	}

  }

  *(C + idx) = temp1;
}




__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * t; 

  //determine the row to process                                              
  A = A + blockIdx.x * blockDim.x + threadIdx.x;
  C = C + blockIdx.x * blockDim.x + threadIdx.x;
  double temp = 0;
                                                                                                                                                                                                                                                                        
  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;
  double r0, r1, r2,r3;//, r4,r5,r6,r7;

  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    __syncthreads();
    B += T;
    
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
	      r0 = *(A + 0 *lda);
	      r1 = *(A + 1 *lda);
	      r2 = *(A + 2 *lda);
	      r3 = *(A + 3 *lda); 
	  }
      
      for (int i = 0; i < t; i++) {
	       temp += cacheA[threadIdx.x + i * T] * cacheB[l - j + i ];
      }
      if (l + t < k) {
      	cacheA[threadIdx.x + 0 * T] = r0;
      	cacheA[threadIdx.x + 1 * T] = r1;
      	cacheA[threadIdx.x + 2 * T] = r2;
      	cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
  }
  *C= temp;
    
}



__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
	extern __shared__ double cache[];

	double * cacheA = cache;

	//determine the row to process                                              
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	A = A + idx;
	double temp = 0;
	                                                                                                                                                                                                                                                                    
	//prefectch A 
	for (int i = 0; i < t; i++){
	cacheA[threadIdx.x + i * T] = *(A + i * lda);
	}
	A += t * lda;
	double r0, r1, r2,r3;//, r4,r5,r6,r7;
    
	for (int l = 0; l < k; l += t){
	  if (l + t < k) {
	      r0 = *(A + 0 *lda);
	      r1 = *(A + 1 *lda);
	      r2 = *(A + 2 *lda);
	      r3 = *(A + 3 *lda); 
	  }
	  
	  for (int i = 0; i < t; i++) {
	       temp += cacheA[threadIdx.x + i * T] * B[l + i];
	  }
	  if (l + t < k) {
	  	cacheA[threadIdx.x + 0 * T] = r0;
	  	cacheA[threadIdx.x + 1 * T] = r1;
	  	cacheA[threadIdx.x + 2 * T] = r2;
	  	cacheA[threadIdx.x + 3 * T] = r3;
	  }
	  A += t * lda;
	}
  	*(C + idx) = temp;
    
}


__global__ void
dgemm_kernel4_3(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * t; 

  //determine the row to process                                              
  A = A + blockIdx.x * blockDim.x + threadIdx.x;
  C = C + blockIdx.x * blockDim.x + threadIdx.x;
  double temp = 0;
                                                                                                                                                                                                                                                                        
  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;
  double r0, r1, r2,r3;//, r4,r5,r6,r7;
  double rb;
  cacheB[threadIdx.x] = *(B + threadIdx.x);

  for (int j = 0; j < k; j += T){ 
  	B += T;
    rb = *(B + threadIdx.x);  
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
	      r0 = *(A + 0 *lda);
	      r1 = *(A + 1 *lda);
	      r2 = *(A + 2 *lda);
	      r3 = *(A + 3 *lda); 
	  }
      
      for (int i = 0; i < t; i++) {
	       temp += cacheA[threadIdx.x + i * T] * cacheB[l - j + i ];
      }
      if (l + t < k) {
      	cacheA[threadIdx.x + 0 * T] = r0;
      	cacheA[threadIdx.x + 1 * T] = r1;
      	cacheA[threadIdx.x + 2 * T] = r2;
      	cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
    __syncthreads();
    cacheB[threadIdx.x] = rb;
    __syncthreads();


  }
  *C = temp;
    
}

