
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>
#include <time.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>
#define TEST_RUN 10 
#define ESP 10e-10
#define PEAK_MEM 900
using namespace std;


void check_cuda_error(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}

void check_C(float * dC, int m, int n, float * checkC) {
  for (int i = 0; i < m * n; i++){
    //cout << i << endl;
    if (fabs(dC[i] - checkC[i]) > ESP){
      cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
      return;
    }
  }
  //cout << "correct" << endl;
}

void output(int m, int n, int k, float min_time, float base, int blocksPerGrid_min, int threadsPerBlock_min, string func) {
  // long long total_bytes = (m * k + k * n * (k / 32)) * sizeof(double);
  long long total_bytes = (m * k + k * n * blocksPerGrid_min) * sizeof(float);
  double total_gb = (double)total_bytes / 1e9;
  total_gb *= TEST_RUN;
  // cout <<func << "("<< blocksPerGrid_min << "*" << threadsPerBlock_min << "): " << min_time << " s" 
  //      <<" ("  << base/min_time <<"x)."
  //      <<" (" << total_gb <<"GB)"
  //      <<" (" << total_gb/min_time <<"GB/s)"<<endl;
  cout << min_time << "," << base/min_time << "," << total_gb/min_time << "," << total_gb/min_time/PEAK_MEM << "\n";
}

/////////////////////////NAIVE/////////////////////////

__global__ void
dgemm_kernel_naive(int m, int n, int k, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float a = 0;
  register float b = 0;
  register float temp = 0;
 
  for (int j = 0; j < n; j++) {
    #pragma unroll 1
    for (int i = 0; i < k; i+=1){
      //load data
      a = *(A + lda * i);
      b = *(B + ldb * j + i);
      //compute
      temp += a * b;
      
    }
    *(C + j * ldc + idx) = temp;
    temp = 0;
  }
  
}


__global__ void
dgemm_kernel_reduce_gld(int m, int n, int k, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float temp1 = 0;
  register float temp2 = 0;
  register float temp3 = 0;
  register float temp4 = 0;
  register float temp5 = 0;
  register float temp6 = 0;
  register float temp7 = 0;
  register float temp8 = 0;

  register float a = 0;

  register float b1 = 0;
  register float b2 = 0;
  register float b3 = 0;
  register float b4 = 0;
  register float b5 = 0;
  register float b6 = 0;
  register float b7 = 0;
  register float b8 = 0;

  #pragma unroll 1
  for (int i = 0; i < k; i+=1){
    //load data
    a = *A;

    b1 = *B;
    b2 = *(B + ldb);
    b3 = *(B + ldb * 2);
    b4 = *(B + ldb * 3);
    b5 = *(B + ldb * 4);
    b6 = *(B + ldb * 5);
    b7 = *(B + ldb * 6);
    b8 = *(B + ldb * 7);

    A += lda;
    B += 1;

    //compute
    temp1 += a * b1;
    temp2 += a * b2;
    temp3 += a * b3;
    temp4 += a * b4;
    temp5 += a * b5;
    temp6 += a * b6;
    temp7 += a * b7;
    temp8 += a * b8;

  }

  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;
  *(C + 4 * ldc + idx) = temp5;
  *(C + 5 * ldc + idx) = temp6;
  *(C + 6 * ldc + idx) = temp7;
  *(C + 7 * ldc + idx) = temp8;
  
}



void test_kernel_naive(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_naive<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V0");
}

void test_kernel_reduce_gld(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_reduce_gld<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V1");
}



/////////////////////////SHARED/////////////////////////
__global__ void
dgemm_kernel_shared(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ float cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float temp1 = 0;
  register float temp2 = 0;
  register float temp3 = 0;
  register float temp4 = 0;
  register float temp5 = 0;
  register float temp6 = 0;
  register float temp7 = 0;
  register float temp8 = 0;

  register float a = 0;

  for (int j = 0; j < k; j += T){
    cache[threadIdx.x * 8 + 0] = *(B + threadIdx.x + ldb * 0);
    cache[threadIdx.x * 8 + 1] = *(B + threadIdx.x + ldb * 1);
    cache[threadIdx.x * 8 + 2] = *(B + threadIdx.x + ldb * 2);
    cache[threadIdx.x * 8 + 3] = *(B + threadIdx.x + ldb * 3);
    
    cache[threadIdx.x * 8 + 4] = *(B + threadIdx.x + ldb * 4);
    cache[threadIdx.x * 8 + 5] = *(B + threadIdx.x + ldb * 5);
    cache[threadIdx.x * 8 + 6] = *(B + threadIdx.x + ldb * 6);
    cache[threadIdx.x * 8 + 7] = *(B + threadIdx.x + ldb * 7);

    __syncthreads();
    B += T;
    for (int i = 0; i < T; i++) {
      a = *(A + (i + j) * lda);
      temp1 += a * cache[i * 8 + 0];
      temp2 += a * cache[i * 8 + 1];
      temp3 += a * cache[i * 8 + 2];
      temp4 += a * cache[i * 8 + 3];

      temp5 += a * cache[i * 8 + 4];
      temp6 += a * cache[i * 8 + 5];
      temp7 += a * cache[i * 8 + 6];
      temp8 += a * cache[i * 8 + 7];
    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

  *(C + 4 * ldc + idx) = temp5;
  *(C + 5 * ldc + idx) = temp6;
  *(C + 6 * ldc + idx) = temp7;
  *(C + 7 * ldc + idx) = temp8;

}


float test_kernel_shared(int m, int n, int k, 
          float * dA, int lda, 
          float * dB, int ldb, 
          float * dC, int ldc,
          float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(512, m); T *= 2) {

    //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_shared<<<blocksPerGrid, threadsPerBlock,  T * sizeof(float) * 8>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V2");
}


///////////////////////A PREFETCH(cache<->register)
__global__ void
dgemm_kernel_prefetch_s2r_16(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{

  extern __shared__ float cache[];
  
  float * cacheA = cache;
  float * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;
  float temp3 = 0;
  float temp4 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  float r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);

      r8 = *(A + 8 *lda);
      r9 = *(A + 9 *lda);
      r10 = *(A + 10 *lda);
      r11 = *(A + 11 *lda);
      r12 = *(A + 12 *lda);
      r13 = *(A + 13 *lda);
      r14 = *(A + 14 *lda);
      r15 = *(A + 15 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 4];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 1];
      temp3 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 2];
      temp4 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 3];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;

      cacheA[threadIdx.x + 8 * T] = r8;
      cacheA[threadIdx.x + 9 * T] = r9;
      cacheA[threadIdx.x + 10 * T] = r10;
      cacheA[threadIdx.x + 11 * T] = r11;
      cacheA[threadIdx.x + 12 * T] = r12;
      cacheA[threadIdx.x + 13 * T] = r13;
      cacheA[threadIdx.x + 14 * T] = r14;
      cacheA[threadIdx.x + 15 * T] = r15;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

}


__global__ void
dgemm_kernel_prefetch_s2r_8(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{

  extern __shared__ float cache[];
  
  float * cacheA = cache;
  float * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;
  float temp3 = 0;
  float temp4 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  float r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 4];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 1];
      temp3 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 2];
      temp4 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 3];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

}


void test_kernel_prefetch(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 8; T <= 16; T *= 2) {
  //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      if (T == 16)
        dgemm_kernel_prefetch_s2r_16<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * T)) * sizeof(float)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      else if (T == 8)
        dgemm_kernel_prefetch_s2r_8<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * T)) * sizeof(float)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-1");
}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, i, j, l (12)
//Double registers: cache, cacheA, cacheB, A, B, C, r0-3, temp1-2 (22)
//Shared mem.: T*2 + T*T (float)
__global__ void
dgemm_kernel_prefetch_s2r_4_16(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ float cache[];
 
  float * cacheA = cache;
  float * cacheB = cache + T * t; //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;
  float temp3 = 0;
  float temp4 = 0;

  #pragma unroll 1
  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;

  float r0, r1, r2, r3;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        r0 = *(A + 0 *lda);
        r1 = *(A + 1 *lda);
        r2 = *(A + 2 *lda);
        r3 = *(A + 3 *lda); 
      }

      #pragma unroll 1
      for (int i = 0; i < t; i++) {
        temp1 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i ];
        temp2 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 1];
        temp3 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 2];
        temp4 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 3];
      }
      if (l + t < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;
    
}

void test_kernel_prefetch2(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++){
      dgemm_kernel_prefetch_s2r_4_16<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * tt)) * sizeof(float)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    output(m, n, k, real_time, base, blocksPerGrid, threadsPerBlock, "V3-2");

}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (float)
//#define t 4
__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ float cacheB[];

  //determine the row to process                                                                                                                                                                                                                          
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  C = C + idx;
  register float temp1 = 0;
  register float temp2 = 0;
  register float temp3 = 0;
  register float temp4 = 0;
  register float temp5 = 0;
  register float temp6 = 0;
  register float temp7 = 0;
  register float temp8 = 0;


  register float nr0, nr1, nr2, nr3;
  register float cr0, cr1, cr2, cr3;

  register float b0, b1, b2, b3, b4, b5, b6, b7;

  //prefectch A 
  cr0 = *A;
  A += lda;
  cr1 = *A;
  A += lda;
  
  cr2 = *A;
  A += lda;
  cr3 = *A;
  A += lda;

 __syncthreads();
   cacheB[threadIdx.x * 8] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 8 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 8 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 8 + 3] = *(B + threadIdx.x + ldb * 3);
    cacheB[threadIdx.x * 8 + 4] = *(B + threadIdx.x + ldb * 4);
    cacheB[threadIdx.x * 8 + 5] = *(B + threadIdx.x + ldb * 5);
    cacheB[threadIdx.x * 8 + 6] = *(B + threadIdx.x + ldb * 6);
    cacheB[threadIdx.x * 8 + 7] = *(B + threadIdx.x + ldb * 7);
  __syncthreads();
  B += T;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 

    b0 = *(B + threadIdx.x);
    b1 = *(B + threadIdx.x + ldb);
    b2 = *(B + threadIdx.x + ldb * 2);
    b3 = *(B + threadIdx.x + ldb * 3);
    b4 = *(B + threadIdx.x + ldb * 4);
    b5 = *(B + threadIdx.x + ldb * 5);
    b6 = *(B + threadIdx.x + ldb * 6);
    b7 = *(B + threadIdx.x + ldb * 7);
 
  B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        nr0 = *A;
        A += lda;
        nr1 = *A;
        A += lda;

        nr2 = *A;
        A += lda;
        nr3 = *A;
        A += lda;
      }

      temp1 += cr0 * cacheB[(l - j)*8 + 0 ];
      temp2 += cr0 * cacheB[(l - j)*8 + 0 + 1];
      temp3 += cr0 * cacheB[(l - j)*8 + 0 + 2];
      temp4 += cr0 * cacheB[(l - j)*8 + 0 + 3];
      temp5 += cr0 * cacheB[(l - j)*8 + 0 + 4];
      temp6 += cr0 * cacheB[(l - j)*8 + 0 + 5];
      temp7 += cr0 * cacheB[(l - j)*8 + 0 + 6];
      temp8 += cr0 * cacheB[(l - j)*8 + 0 + 7];

      temp1 += cr1 * cacheB[(l - j)*8 + 1 ];
      temp2 += cr1 * cacheB[(l - j)*8 + 1 + 1];
      temp3 += cr1 * cacheB[(l - j)*8 + 1 + 2];
      temp4 += cr1 * cacheB[(l - j)*8 + 1 + 3];
      temp5 += cr1 * cacheB[(l - j)*8 + 1 + 3];
      temp6 += cr1 * cacheB[(l - j)*8 + 1 + 3];
      temp7 += cr1 * cacheB[(l - j)*8 + 1 + 3];
      temp8 += cr1 * cacheB[(l - j)*8 + 1 + 3];

      temp1 += cr2 * cacheB[(l - j) * 8 + 2 ];
      temp2 += cr2 * cacheB[(l - j) * 8 + 2 + 1];
      temp3 += cr2 * cacheB[(l - j) * 8 + 2 + 2];
      temp4 += cr2 * cacheB[(l - j) * 8 + 2 + 3];
      temp5 += cr2 * cacheB[(l - j) * 8 + 2 + 4];
      temp6 += cr2 * cacheB[(l - j) * 8 + 2 + 5];
      temp7 += cr2 * cacheB[(l - j) * 8 + 2 + 6];
      temp8 += cr2 * cacheB[(l - j) * 8 + 2 + 7];

      temp1 += cr3 * cacheB[(l - j)*8 + 3 ];
      temp2 += cr3 * cacheB[(l - j)*8 + 3 + 1];
      temp3 += cr3 * cacheB[(l - j)*8 + 3 + 2];
      temp4 += cr3 * cacheB[(l - j)*8 + 3 + 3];
      temp5 += cr3 * cacheB[(l - j)*8 + 3 + 4];
      temp6 += cr3 * cacheB[(l - j)*8 + 3 + 5];
      temp7 += cr3 * cacheB[(l - j)*8 + 3 + 6];
      temp8 += cr3 * cacheB[(l - j)*8 + 3 + 7];

      if (l + t < k) {
        cr0 = nr0;
        cr1 = nr1;
        cr2 = nr2;
        cr3 = nr3;
      }
    }
    __syncthreads();
    cacheB[threadIdx.x * 8] = b0;
    cacheB[threadIdx.x * 8 + 1] = b1;
    cacheB[threadIdx.x * 8 + 2] = b2;
    cacheB[threadIdx.x * 8 + 3] = b3;
    cacheB[threadIdx.x * 8 + 4] = b4;
    cacheB[threadIdx.x * 8 + 5] = b5;
    cacheB[threadIdx.x * 8 + 6] = b6;
    cacheB[threadIdx.x * 8 + 7] = b7;
    __syncthreads();


  }
  *C = temp1;
  *(C + ldc) = temp2;
  *(C + ldc * 2) = temp3;
  *(C + ldc * 3) = temp4;
  *(C + ldc * 4) = temp5;
  *(C + ldc * 5) = temp6;
  *(C + ldc * 6) = temp7;
  *(C + ldc * 7) = temp8;
    
}





//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (float)

__global__ void
dgemm_kernel4_3(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
                                                                                                                                                                                                                
  // //determine the row to process                                                                                                                                                                                                                          
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // A = A + idx;
  // C = C + idx;
  // register float temp1 = 0;
  // register float temp2 = 0;
  // register float temp3 = 0;
  // register float temp4 = 0;

  // register float nr0, nr1;//, nr2, nr3;
  // register float cr0, cr1;//, cr2, cr3;

  // register float nb00, nb01, nb02, nb03;//, nb10, nb11, nb12, nb13;
  // register float cb00, cb01, cb02, cb03;//, cb10, cb11, cb12, cb13;

  // //prefectch A 
  // cr0 = *A;
  // A += lda;
  // cr1 = *A;
  // A += lda;
  
  // // cr2 = *A;
  // // A += lda;
  // // cr3 = *A;
  // // A += lda;

  // cb00 = *B;
  // cb01 = *(B + ldb);
  // cb02 = *(B + ldb * 2);
  // cb03 = *(B + ldb * 3);
  // B += 1;
  // // cb10 = *B;
  // // cb11 = *(B + ldb);
  // // cb12 = *(B + ldb * 2);
  // // cb13 = *(B + ldb * 3);
  // // B += 1;


  // #pragma unroll 1
  // for (int i = 0; i < k; i += t){ 
  //     if (i + t < k) {
  //       nr0 = *A;
  //       A += lda;
  //       nr1 = *A;
  //       A += lda;

  //       // nr2 = *A;
  //       // A += lda;
  //       // nr3 = *A;
  //       // A += lda;
  //     }

  //     nb00 = *B;
  //     nb01 = *(B + ldb);
  //     // nb02 = *(B + ldb * 2);
  //     // nb03 = *(B + ldb * 3);
  //     B += 1;
  //     // nb10 = *B;
  //     // nb11 = *(B + ldb);
  //     // nb12 = *(B + ldb * 2);
  //     // nb13 = *(B + ldb * 3);
  //     // B += 1;

  //     temp1 += cr0 * cb00;
  //     temp2 += cr0 * cb01;
  //     temp3 += cr0 * cb02;
  //     temp4 += cr0 * cb03;

  //     // temp1 += cr1 * cb10;
  //     // temp2 += cr1 * cb11;
  //     // temp3 += cr1 * cb12;
  //     // temp4 += cr1 * cb13;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb02 = nb02;
  //     cb03 = nb03;

  //     // cb10 = nb10;
  //     // cb11 = nb11;
  //     // cb12 = nb12;
  //     // cb13 = nb13;

  //     nb00 = *B;
  //     nb01 = *(B + ldb);
  //     nb02 = *(B + ldb * 2);
  //     nb03 = *(B + ldb * 3);
  //     B += 1;

  //     temp1 += cr1 * cb00;
  //     temp2 += cr1 * cb01;
  //     temp3 += cr1 * cb02;
  //     temp4 += cr1 * cb03;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb02 = nb02;
  //     cb03 = nb03;



  //     // nb00 = *B;
  //     // nb01 = *(B + ldb);
  //     // nb02 = *(B + ldb * 2);
  //     // nb03 = *(B + ldb * 3);
  //     // B += 1;

  //     // temp1 += cr2 * cb00;
  //     // temp2 += cr2 * cb01;
  //     // temp3 += cr2 * cb02;
  //     // temp4 += cr2 * cb03;

  //     // cb00 = nb00;
  //     // cb01 = nb01;
  //     // cb02 = nb02;
  //     // cb03 = nb03;




  //     // if (i + t < k) {
  //     //   nb00 = *B;
  //     //   nb01 = *(B + ldb);
  //     //   nb02 = *(B + ldb * 2);
  //     //   nb03 = *(B + ldb * 3);
  //     //    B += 1;
  //     //   // nb10 = *B;
  //     //   // nb11 = *(B + ldb);
  //     //   // nb12 = *(B + ldb * 2);
  //     //   // nb13 = *(B + ldb * 3);
  //     //   // B += 1;
  //     // }

  //     // temp1 += cr3 * cb00;
  //     // temp2 += cr3 * cb01;
  //     // temp3 += cr3 * cb02;
  //     // temp4 += cr3 * cb03;
  //     // // temp1 += cr4 * cb10;
  //     // // temp2 += cr4 * cb11;

  //     // cb00 = nb00;
  //     // cb01 = nb01;
  //     // cb02 = nb02;
  //     // cb03 = nb03;

  //     // // cb10 = nb10;
  //     // // cb11 = nb11;
  //     // // cb12 = nb12;
  //     // // cb13 = nb13;
    

  //     if (i + t < k) {
  //       cr0 = nr0;
  //       cr1 = nr1;
  //       // cr2 = nr2;
  //       // cr3 = nr3;
  //     }
  // }
  // *C = temp1;
  // *(C + ldc) = temp2;
  // *(C + ldc * 2) = temp3;
  // *(C + ldc * 3) = temp4;
    
}


float test_kernel_prefetch3(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(m, 512); T*=2) {
  //int T = 128;
  int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_2<<<blocksPerGrid, threadsPerBlock, (T * 8) * sizeof(float)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-3");
}



float test_kernel_prefetch4(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(m, 1024); T*=2) {
    //int T = 128;
    int tt = 2;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_3<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-4");
}




float test_cublas_mm(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc);


void test(int m, int k, int c);

int main(int argc, char *argv[]){
  for (int i = 10240; i <= 40960; i += 1024){
  //int i = 1024;
    // cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 2 << ")" << endl;
    test(i, i, atoi(argv[1]));
  }
}

void test(int m, int k, int c){
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 8;
    //int k = 20480;
    float * A = new float[m * k];
    float * B = new float[n * k];
    float * C = new float[m * n];
    float * checkC = new float[m * n];     

    for (int i = 0; i < m * k; i++){
    	A[i] = i;
    }

    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    float * dA;
    hipMalloc(&dA, m * k * sizeof(float));
    int lda = m;

    float * dB; 
    hipMalloc(&dB,  n * k * sizeof(float));
    int ldb = k;

    float * dC;
    hipMalloc(&dC, m * n * sizeof(float));
    int ldc = m;

    float * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(float));

    hipMemcpy(dA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(float), hipMemcpyHostToDevice);
    
    float base;

    base = test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
    if (c == -1) cout << base << endl;
    if (c == 0) test_kernel_naive(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 1) test_kernel_reduce_gld(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 2) test_kernel_shared(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch2(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 3) test_kernel_prefetch3(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch4(m, n, k, dA, lda, dB, ldb, dC, ldc, base);


    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}



float test_cublas_mm(int m, int n, int k, 
         float * dA, int lda, 
         float * dB, int ldb, 
         float * dC, int ldc){

    float one = 1;
    float zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &one, dA, lda, dB, ldb, &zero, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    // cout <<"Runing time of culasdgemm:" << real_time <<" s." << endl;

    return real_time;
}






























