
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>
#include <time.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>
#define TEST_RUN 1 
#define ESP 10e-10
#define PEAK_MEM 900
using namespace std;


void check_cuda_error(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}

void check_C(double * dC, int m, int n, double * checkC) {
  for (int i = 0; i < m * n; i++){
    //cout << i << endl;
    if (fabs(dC[i] - checkC[i]) > ESP){
      cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
      return;
    }
  }
  //cout << "correct" << endl;
}

void output(int m, int n, int k, float min_time, float base, int blocksPerGrid_min, int threadsPerBlock_min, string func) {
  // long long total_bytes = (m * k + k * n * (k / 32)) * sizeof(double);
  long long total_bytes = (m * k + k * n * blocksPerGrid_min) * sizeof(double);
  double total_gb = (double)total_bytes / 1e9;
  total_gb *= TEST_RUN;
  // cout <<func << "("<< blocksPerGrid_min << "*" << threadsPerBlock_min << "): " << min_time << " s" 
  //      <<" ("  << base/min_time <<"x)."
  //      <<" (" << total_gb <<"GB)"
  //      <<" (" << total_gb/min_time <<"GB/s)"<<endl;
  cout << min_time << "," << base/min_time << "," << total_gb/min_time << "," << total_gb/min_time/PEAK_MEM << "\n";

}

/////////////////////////NAIVE/////////////////////////

__global__ void
dgemm_kernel_naive(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register double a = 0;
  register double b = 0;
  register double temp = 0;
 
  for (int j = 0; j < n; j++) {
    #pragma unroll 1
    for (int i = 0; i < k; i+=1){
      //load data
      a = *(A + lda * i);
      b = *(B + ldb * j + i);
      //compute
      temp += a * b;
      
    }
    *(C + j * ldc + idx) = temp;
    temp = 0;
  }
  
}


__global__ void
dgemm_kernel_reduce_gld(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register double temp1 = 0;
  register double temp2 = 0;
  register double temp3 = 0;
  register double temp4 = 0;
  register double temp5 = 0;
  register double temp6 = 0;
  register double temp7 = 0;
  register double temp8 = 0;
  register double temp9 = 0;
  register double temp10 = 0;
  register double temp11 = 0;
  register double temp12 = 0;
  register double temp13 = 0;
  register double temp14 = 0;
  register double temp15 = 0;
  register double temp16 = 0;

  register double a = 0;

  register double b1 = 0;
  register double b2 = 0;
  register double b3 = 0;
  register double b4 = 0;
  register double b5 = 0;
  register double b6 = 0;
  register double b7 = 0;
  register double b8 = 0;
  register double b9 = 0;
  register double b10 = 0;
  register double b11 = 0;
  register double b12 = 0;
  register double b13 = 0;
  register double b14 = 0;
  register double b15 = 0;
  register double b16 = 0;

  #pragma unroll 1
  for (int i = 0; i < k; i+=1){
    //load data
    a = *A;
    
    b1 = *B;
    b2 = *(B + ldb);
    b3 = *(B + ldb * 2);
    b4 = *(B + ldb * 3);
    b5 = *(B + ldb * 4);
    b6 = *(B + ldb * 5);
    b7 = *(B + ldb * 6);
    b8 = *(B + ldb * 7);
    b9 = *(B + ldb * 8);
    b10 = *(B + ldb * 9);
    b11 = *(B + ldb * 10);
    b12 = *(B + ldb * 11);
    b13 = *(B + ldb * 12);
    b14 = *(B + ldb * 13);
    b15 = *(B + ldb * 14);
    b16 = *(B + ldb * 15);


    A += lda;
    B += 1;

    //compute
    temp1 += a * b1;
    temp2 += a * b2;
    temp3 += a * b3;
    temp4 += a * b4;
    temp5 += a * b5;
    temp6 += a * b6;
    temp7 += a * b7;
    temp8 += a * b8;
    temp9 += a * b9;
    temp10 += a * b10;
    temp11 += a * b11;
    temp12 += a * b12;
    temp13 += a * b13;
    temp14 += a * b14;
    temp15 += a * b15;
    temp16 += a * b16;

  }

  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;
  *(C + 4 * ldc + idx) = temp5;
  *(C + 5 * ldc + idx) = temp6;
  *(C + 6 * ldc + idx) = temp7;
  *(C + 7 * ldc + idx) = temp8;
  *(C + 8 * ldc + idx) = temp9;
  *(C + 9 * ldc + idx) = temp10;
  *(C + 10 * ldc + idx) = temp11;
  *(C + 11 * ldc + idx) = temp12;
  *(C + 12 * ldc + idx) = temp13;
  *(C + 13 * ldc + idx) = temp14;
  *(C + 14 * ldc + idx) = temp15;
  *(C + 15 * ldc + idx) = temp16;
  
}


void test_kernel_naive(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_naive<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V0");

}

void test_kernel_reduce_gld(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_reduce_gld<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V1");
}


/////////////////////////SHARED/////////////////////////
__global__ void
dgemm_kernel_shared(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register double temp1 = 0;
  register double temp2 = 0;
  register double temp3 = 0;
  register double temp4 = 0;
  register double temp5 = 0;
  register double temp6 = 0;
  register double temp7 = 0;
  register double temp8 = 0;
  register double temp9 = 0;
  register double temp10 = 0;
  register double temp11 = 0;
  register double temp12 = 0;
  register double temp13 = 0;
  register double temp14 = 0;
  register double temp15 = 0;
  register double temp16 = 0;

  register double a = 0;

  for (int j = 0; j < k; j += T){
    cache[threadIdx.x * 16 + 0] = *(B + threadIdx.x + ldb * 0);
    cache[threadIdx.x * 16 + 1] = *(B + threadIdx.x + ldb * 1);
    cache[threadIdx.x * 16 + 2] = *(B + threadIdx.x + ldb * 2);
    cache[threadIdx.x * 16 + 3] = *(B + threadIdx.x + ldb * 3);
    
    cache[threadIdx.x * 16 + 4] = *(B + threadIdx.x + ldb * 4);
    cache[threadIdx.x * 16 + 5] = *(B + threadIdx.x + ldb * 5);
    cache[threadIdx.x * 16 + 6] = *(B + threadIdx.x + ldb * 6);
    cache[threadIdx.x * 16 + 7] = *(B + threadIdx.x + ldb * 7);
  
    cache[threadIdx.x * 16 + 8] = *(B + threadIdx.x + ldb * 8);
    cache[threadIdx.x * 16 + 9] = *(B + threadIdx.x + ldb * 9);
    cache[threadIdx.x * 16 + 10] = *(B + threadIdx.x + ldb * 10);
    cache[threadIdx.x * 16 + 11] = *(B + threadIdx.x + ldb * 11);
    
    cache[threadIdx.x * 16 + 12] = *(B + threadIdx.x + ldb * 12);
    cache[threadIdx.x * 16 + 13] = *(B + threadIdx.x + ldb * 13);
    cache[threadIdx.x * 16 + 14] = *(B + threadIdx.x + ldb * 14);
    cache[threadIdx.x * 16 + 15] = *(B + threadIdx.x + ldb * 15);
    __syncthreads();
    B += T;
    for (int i = 0; i < T; i++) {
      a = *(A + (i + j) * lda);

      temp1 += a * cache[i * 16 + 0];
      temp2 += a * cache[i * 16 + 1];
      temp3 += a * cache[i * 16 + 2];
      temp4 += a * cache[i * 16 + 3];

      temp5 += a * cache[i * 16 + 4];
      temp6 += a * cache[i * 16 + 5];
      temp7 += a * cache[i * 16 + 6];
      temp8 += a * cache[i * 16 + 7];

      temp9 += a * cache[i * 16 + 8];
      temp10 += a * cache[i * 16 + 9];
      temp11 += a * cache[i * 16 + 10];
      temp12 += a * cache[i * 16 + 11];

      temp13 += a * cache[i * 16 + 12];
      temp14 += a * cache[i * 16 + 13];
      temp15 += a * cache[i * 16 + 14];
      temp16 += a * cache[i * 16 + 15];

    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

  *(C + 4 * ldc + idx) = temp5;
  *(C + 5 * ldc + idx) = temp6;
  *(C + 6 * ldc + idx) = temp7;
  *(C + 7 * ldc + idx) = temp8;

  *(C + 8 * ldc + idx) = temp9;
  *(C + 9 * ldc + idx) = temp10;
  *(C + 10 * ldc + idx) = temp11;
  *(C + 11 * ldc + idx) = temp12;

  *(C + 12 * ldc + idx) = temp13;
  *(C + 13 * ldc + idx) = temp14;
  *(C + 14 * ldc + idx) = temp15;
  *(C + 15 * ldc + idx) = temp16;

}


float test_kernel_shared(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc,
          float base){
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(256, m); T *= 2) { // T <= 256 limited by shared memory per thread block

    //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_shared<<<blocksPerGrid, threadsPerBlock,  T * sizeof(double) * 16>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V2");
}

///////////////////////A PREFETCH(cache<->register)
__global__ void
dgemm_kernel_prefetch_s2r_16(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{

  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;
  double temp3 = 0;
  double temp4 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  double r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);

      r8 = *(A + 8 *lda);
      r9 = *(A + 9 *lda);
      r10 = *(A + 10 *lda);
      r11 = *(A + 11 *lda);
      r12 = *(A + 12 *lda);
      r13 = *(A + 13 *lda);
      r14 = *(A + 14 *lda);
      r15 = *(A + 15 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 4];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 1];
      temp3 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 2];
      temp4 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 3];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;

      cacheA[threadIdx.x + 8 * T] = r8;
      cacheA[threadIdx.x + 9 * T] = r9;
      cacheA[threadIdx.x + 10 * T] = r10;
      cacheA[threadIdx.x + 11 * T] = r11;
      cacheA[threadIdx.x + 12 * T] = r12;
      cacheA[threadIdx.x + 13 * T] = r13;
      cacheA[threadIdx.x + 14 * T] = r14;
      cacheA[threadIdx.x + 15 * T] = r15;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

}


__global__ void
dgemm_kernel_prefetch_s2r_8(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{

  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;
  double temp3 = 0;
  double temp4 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  double r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 4];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 1];
      temp3 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 2];
      temp4 += cacheA[threadIdx.x +i * T] * cacheB[i * 4 + 3];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;

}


void test_kernel_prefetch(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 8; T <= 16; T *= 2) {
    //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      if (T == 16)
        dgemm_kernel_prefetch_s2r_16<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      else if (T == 8)
        dgemm_kernel_prefetch_s2r_8<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-1");
}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, i, j, l (12)
//Double registers: cache, cacheA, cacheB, A, B, C, r0-3, temp1-2 (22)
//Shared mem.: T*2 + T*T (double)
__global__ void
dgemm_kernel_prefetch_s2r_4_16(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * t; //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;
  double temp3 = 0;
  double temp4 = 0;

  #pragma unroll 1
  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;

  double r0, r1, r2, r3;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 4] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 4 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 4 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 4 + 3] = *(B + threadIdx.x + ldb * 3);
    __syncthreads();
    B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        r0 = *(A + 0 *lda);
        r1 = *(A + 1 *lda);
        r2 = *(A + 2 *lda);
        r3 = *(A + 3 *lda); 
      }

      #pragma unroll 1
      for (int i = 0; i < t; i++) {
        temp1 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i ];
        temp2 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 1];
        temp3 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 2];
        temp4 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 3];
      }
      if (l + t < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  *(C + 2 * ldc + idx) = temp3;
  *(C + 3 * ldc + idx) = temp4;
    
}

void test_kernel_prefetch2(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++){
      dgemm_kernel_prefetch_s2r_4_16<<<blocksPerGrid, threadsPerBlock, ((T * 4) + (T * tt)) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    output(m, n, k, real_time, base, blocksPerGrid, threadsPerBlock, "V3-2");

}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (double)
//#define t 4
__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cacheB[];

  //determine the row to process                                                                                                                                                                                                                          
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  C = C + idx;
  register double temp1 = 0;
  register double temp2 = 0;
  register double temp3 = 0;
  register double temp4 = 0;
  register double temp5 = 0;
  register double temp6 = 0;
  register double temp7 = 0;
  register double temp8 = 0;
  register double temp9 = 0;
  register double temp10 = 0;
  register double temp11 = 0;
  register double temp12 = 0;
  register double temp13 = 0;
  register double temp14 = 0;
  register double temp15 = 0;
  register double temp16 = 0;


  register double nr0, nr1, nr2, nr3;
  register double cr0, cr1, cr2, cr3;

  //prefectch A 
  cr0 = *A;
  A += lda;
  cr1 = *A;
  A += lda;
  
  cr2 = *A;
  A += lda;
  cr3 = *A;
  A += lda;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 

    __syncthreads();
    cacheB[threadIdx.x * 16] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 16 + 1] = *(B + threadIdx.x + ldb);
    cacheB[threadIdx.x * 16 + 2] = *(B + threadIdx.x + ldb * 2);
    cacheB[threadIdx.x * 16 + 3] = *(B + threadIdx.x + ldb * 3);
    cacheB[threadIdx.x * 16 + 4] = *(B + threadIdx.x + ldb * 4);
    cacheB[threadIdx.x * 16 + 5] = *(B + threadIdx.x + ldb * 5);
    cacheB[threadIdx.x * 16 + 6] = *(B + threadIdx.x + ldb * 6);
    cacheB[threadIdx.x * 16 + 7] = *(B + threadIdx.x + ldb * 7);
    cacheB[threadIdx.x * 16 + 8] = *(B + threadIdx.x + ldb * 8);
    cacheB[threadIdx.x * 16 + 9] = *(B + threadIdx.x + ldb * 9);
    cacheB[threadIdx.x * 16 + 10] = *(B + threadIdx.x + ldb * 10);
    cacheB[threadIdx.x * 16 + 11] = *(B + threadIdx.x + ldb * 11);
    cacheB[threadIdx.x * 16 + 12] = *(B + threadIdx.x + ldb * 12);
    cacheB[threadIdx.x * 16 + 13] = *(B + threadIdx.x + ldb * 13);
    cacheB[threadIdx.x * 16 + 14] = *(B + threadIdx.x + ldb * 14);
    cacheB[threadIdx.x * 16 + 15] = *(B + threadIdx.x + ldb * 15);
    __syncthreads();
    B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        nr0 = *A;
        A += lda;
        nr1 = *A;
        A += lda;

        nr2 = *A;
        A += lda;
        nr3 = *A;
        A += lda;
      }

      temp1 += cr0 * cacheB[l - j + 0 ];
      temp2 += cr0 * cacheB[l - j + 0 + 1];
      temp3 += cr0 * cacheB[l - j + 0 + 2];
      temp4 += cr0 * cacheB[l - j + 0 + 3];
      temp5 += cr0 * cacheB[l - j + 0 + 4];
      temp6 += cr0 * cacheB[l - j + 0 + 5];
      temp7 += cr0 * cacheB[l - j + 0 + 6];
      temp8 += cr0 * cacheB[l - j + 0 + 7];
      temp9 += cr0 * cacheB[l - j + 0  + 8];
      temp10 += cr0 * cacheB[l - j + 0 + 9];
      temp11 += cr0 * cacheB[l - j + 0 + 10];
      temp12 += cr0 * cacheB[l - j + 0 + 11];
      temp13 += cr0 * cacheB[l - j + 0 + 12];
      temp14 += cr0 * cacheB[l - j + 0 + 13];
      temp15 += cr0 * cacheB[l - j + 0 + 14];
      temp16 += cr0 * cacheB[l - j + 0 + 15];

      temp1 += cr1 * cacheB[l - j + 1 ];
      temp2 += cr1 * cacheB[l - j + 1 + 1];
      temp3 += cr1 * cacheB[l - j + 1 + 2];
      temp4 += cr1 * cacheB[l - j + 1 + 3];
      temp5 += cr1 * cacheB[l - j + 1 + 4];
      temp6 += cr1 * cacheB[l - j + 1 + 5];
      temp7 += cr1 * cacheB[l - j + 1 + 6];
      temp8 += cr1 * cacheB[l - j + 1 + 7];
      temp9 += cr1 * cacheB[l - j + 1  + 8];
      temp10 += cr1 * cacheB[l - j + 1 + 9];
      temp11 += cr1 * cacheB[l - j + 1 + 10];
      temp12 += cr1 * cacheB[l - j + 1 + 11];
      temp13 += cr1 * cacheB[l - j + 1 + 12];
      temp14 += cr1 * cacheB[l - j + 1 + 13];
      temp15 += cr1 * cacheB[l - j + 1 + 14];
      temp16 += cr1 * cacheB[l - j + 1 + 15];

      temp1 += cr2 * cacheB[l - j + 2 ];
      temp2 += cr2 * cacheB[l - j + 2 + 1];
      temp3 += cr2 * cacheB[l - j + 2 + 2];
      temp4 += cr2 * cacheB[l - j + 2 + 3];
      temp5 += cr2 * cacheB[l - j + 2 + 4];
      temp6 += cr2 * cacheB[l - j + 2 + 5];
      temp7 += cr2 * cacheB[l - j + 2 + 6];
      temp8 += cr2 * cacheB[l - j + 2 + 7];
      temp9 += cr2 * cacheB[l - j + 2 + 8];
      temp10 += cr2 * cacheB[l - j + 2 + 9];
      temp11 += cr2 * cacheB[l - j + 2 + 10];
      temp12 += cr2 * cacheB[l - j + 2 + 11];
      temp13 += cr2 * cacheB[l - j + 2 + 12];
      temp14 += cr2 * cacheB[l - j + 2 + 13];
      temp15 += cr2 * cacheB[l - j + 2 + 14];
      temp16 += cr2 * cacheB[l - j + 2 + 15];

      temp1 += cr3 * cacheB[l - j + 3 ];
      temp2 += cr3 * cacheB[l - j + 3 + 1];
      temp3 += cr3 * cacheB[l - j + 3 + 2];
      temp4 += cr3 * cacheB[l - j + 3 + 3];
      temp5 += cr3 * cacheB[l - j + 3 + 4];
      temp6 += cr3 * cacheB[l - j + 3 + 5];
      temp7 += cr3 * cacheB[l - j + 3 + 6];
      temp8 += cr3 * cacheB[l - j + 3 + 7];
      temp9 += cr3 * cacheB[l - j + 3 + 8 ];
      temp10 += cr3 * cacheB[l - j + 3 + 9];
      temp11 += cr3 * cacheB[l - j + 3 + 10];
      temp12 += cr3 * cacheB[l - j + 3 + 11];
      temp13 += cr3 * cacheB[l - j + 3 + 12];
      temp14 += cr3 * cacheB[l - j + 3 + 13];
      temp15 += cr3 * cacheB[l - j + 3 + 14];
      temp16 += cr3 * cacheB[l - j + 3 + 15];

      if (l + t < k) {
        cr0 = nr0;
        cr1 = nr1;
        cr2 = nr2;
        cr3 = nr3;
      }
    }
  }
  *C = temp1;
  *(C + ldc) = temp2;
  *(C + ldc * 2) = temp3;
  *(C + ldc * 3) = temp4;
  *(C + ldc * 4) = temp5;
  *(C + ldc * 5) = temp6;
  *(C + ldc * 6) = temp7;
  *(C + ldc * 7) = temp8;
  *(C + ldc * 8) = temp9;
  *(C + ldc * 9) = temp10;
  *(C + ldc * 10) = temp11;
  *(C + ldc * 11) = temp12;
  *(C + ldc * 12) = temp13;
  *(C + ldc * 13) = temp14;
  *(C + ldc * 14) = temp15;
  *(C + ldc * 15) = temp16;

    
}





//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (double)

__global__ void
dgemm_kernel4_3(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
                                                                                                                                                                                                                
  // //determine the row to process                                                                                                                                                                                                                          
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // A = A + idx;
  // C = C + idx;
  // register double temp1 = 0;
  // register double temp2 = 0;
  // register double temp3 = 0;
  // register double temp4 = 0;

  // register double nr0, nr1;//, nr2, nr3;
  // register double cr0, cr1;//, cr2, cr3;

  // register double nb00, nb01, nb02, nb03;//, nb10, nb11, nb12, nb13;
  // register double cb00, cb01, cb02, cb03;//, cb10, cb11, cb12, cb13;

  // //prefectch A 
  // cr0 = *A;
  // A += lda;
  // cr1 = *A;
  // A += lda;
  
  // // cr2 = *A;
  // // A += lda;
  // // cr3 = *A;
  // // A += lda;

  // cb00 = *B;
  // cb01 = *(B + ldb);
  // cb02 = *(B + ldb * 2);
  // cb03 = *(B + ldb * 3);
  // B += 1;
  // // cb10 = *B;
  // // cb11 = *(B + ldb);
  // // cb12 = *(B + ldb * 2);
  // // cb13 = *(B + ldb * 3);
  // // B += 1;


  // #pragma unroll 1
  // for (int i = 0; i < k; i += t){ 
  //     if (i + t < k) {
  //       nr0 = *A;
  //       A += lda;
  //       nr1 = *A;
  //       A += lda;

  //       // nr2 = *A;
  //       // A += lda;
  //       // nr3 = *A;
  //       // A += lda;
  //     }

  //     nb00 = *B;
  //     nb01 = *(B + ldb);
  //     // nb02 = *(B + ldb * 2);
  //     // nb03 = *(B + ldb * 3);
  //     B += 1;
  //     // nb10 = *B;
  //     // nb11 = *(B + ldb);
  //     // nb12 = *(B + ldb * 2);
  //     // nb13 = *(B + ldb * 3);
  //     // B += 1;

  //     temp1 += cr0 * cb00;
  //     temp2 += cr0 * cb01;
  //     temp3 += cr0 * cb02;
  //     temp4 += cr0 * cb03;

  //     // temp1 += cr1 * cb10;
  //     // temp2 += cr1 * cb11;
  //     // temp3 += cr1 * cb12;
  //     // temp4 += cr1 * cb13;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb02 = nb02;
  //     cb03 = nb03;

  //     // cb10 = nb10;
  //     // cb11 = nb11;
  //     // cb12 = nb12;
  //     // cb13 = nb13;

  //     nb00 = *B;
  //     nb01 = *(B + ldb);
  //     nb02 = *(B + ldb * 2);
  //     nb03 = *(B + ldb * 3);
  //     B += 1;

  //     temp1 += cr1 * cb00;
  //     temp2 += cr1 * cb01;
  //     temp3 += cr1 * cb02;
  //     temp4 += cr1 * cb03;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb02 = nb02;
  //     cb03 = nb03;



  //     // nb00 = *B;
  //     // nb01 = *(B + ldb);
  //     // nb02 = *(B + ldb * 2);
  //     // nb03 = *(B + ldb * 3);
  //     // B += 1;

  //     // temp1 += cr2 * cb00;
  //     // temp2 += cr2 * cb01;
  //     // temp3 += cr2 * cb02;
  //     // temp4 += cr2 * cb03;

  //     // cb00 = nb00;
  //     // cb01 = nb01;
  //     // cb02 = nb02;
  //     // cb03 = nb03;




  //     // if (i + t < k) {
  //     //   nb00 = *B;
  //     //   nb01 = *(B + ldb);
  //     //   nb02 = *(B + ldb * 2);
  //     //   nb03 = *(B + ldb * 3);
  //     //    B += 1;
  //     //   // nb10 = *B;
  //     //   // nb11 = *(B + ldb);
  //     //   // nb12 = *(B + ldb * 2);
  //     //   // nb13 = *(B + ldb * 3);
  //     //   // B += 1;
  //     // }

  //     // temp1 += cr3 * cb00;
  //     // temp2 += cr3 * cb01;
  //     // temp3 += cr3 * cb02;
  //     // temp4 += cr3 * cb03;
  //     // // temp1 += cr4 * cb10;
  //     // // temp2 += cr4 * cb11;

  //     // cb00 = nb00;
  //     // cb01 = nb01;
  //     // cb02 = nb02;
  //     // cb03 = nb03;

  //     // // cb10 = nb10;
  //     // // cb11 = nb11;
  //     // // cb12 = nb12;
  //     // // cb13 = nb13;
    

  //     if (i + t < k) {
  //       cr0 = nr0;
  //       cr1 = nr1;
  //       // cr2 = nr2;
  //       // cr3 = nr3;
  //     }
  // }
  // *C = temp1;
  // *(C + ldc) = temp2;
  // *(C + ldc * 2) = temp3;
  // *(C + ldc * 3) = temp4;
    
}


float test_kernel_prefetch3(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(m, 256); T*=2) {
    //int T = 128;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_2<<<blocksPerGrid, threadsPerBlock, (T * 16) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-3");
}



float test_kernel_prefetch4(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;

  for (int T = 16; T <= min(m, 1024); T*=2) {
  //int T = 128;
  int tt = 2;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_3<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-4");

}




float test_cublas_mm(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc);


void test(int m, int k, int c);


int main(int argc, char *argv[]){
  for (int i = 10240; i <= 40960; i += 1024){
  //int i = 1024;
    // cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 2 << ")" << endl;
    test(i, i, atoi(argv[1]));
  }
}

void test(int m, int k, int c){
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //m = 16;
    int n = 16;
    //int k = 20480;
    double * A = new double[m * k];
    double * B = new double[n * k];
    double * C = new double[m * n];
    double * checkC = new double[m * n];     

    for (int i = 0; i < m * k; i++){
    	A[i] = i;
    }

    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    double * dA;
    hipMalloc(&dA, m * k * sizeof(double));
    int lda = m;

    double * dB; 
    hipMalloc(&dB,  n * k * sizeof(double));
    int ldb = k;

    double * dC;
    hipMalloc(&dC, m * n * sizeof(double));
    int ldc = m;

    double * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(double));

    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    
    float base;

    base = test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
    if (c == -1) cout << base << endl;
    if (c == 0) test_kernel_naive(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 1) test_kernel_reduce_gld(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 2) test_kernel_shared(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch2(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 3) test_kernel_prefetch3(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch4(m, n, k, dA, lda, dB, ldb, dC, ldc, base);

    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}



float test_cublas_mm(int m, int n, int k, 
         double * dA, int lda, 
         double * dB, int ldb, 
         double * dC, int ldc){

    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &one, dA, lda, dB, ldb, &zero, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    // cout <<"Runing time of culasdgemm:" << real_time <<" s." << endl;

    return real_time;
}






























