
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>
#include <time.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>
#define TEST_RUN 10 
#define ESP 10e-10
#define PEAK_MEM 900
using namespace std;


void check_cuda_error(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}

void check_C(float * dC, int m, int n, float * checkC) {
  for (int i = 0; i < m * n; i++){
    //cout << i << endl;
    if (fabs(dC[i] - checkC[i]) > ESP){
      cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
      return;
    }
  }
  cout << "correct" << endl;
}

void output(int m, int n, int k, float min_time, float base, int blocksPerGrid_min, int threadsPerBlock_min, string func) {
  // long long total_bytes = (m * k + k * n * (k / 32)) * sizeof(double);
  long long total_bytes = (m * k + k * n * blocksPerGrid_min) * sizeof(float);
  double total_gb = (double)total_bytes / 1e9;
  total_gb *= TEST_RUN;
  // cout <<func << "("<< blocksPerGrid_min << "*" << threadsPerBlock_min << "): " << min_time << " s" 
  //      <<" ("  << base/min_time <<"x)."
  //      <<" (" << total_gb <<"GB)"
  //      <<" (" << total_gb/min_time <<"GB/s)"<<endl;
  cout << min_time << "," << base/min_time << "," << total_gb/min_time << "," << total_gb/min_time/PEAK_MEM << "\n";

}


/////////////////////////NAIVE/////////////////////////

__global__ void
dgemm_kernel_naive(int m, int n, int k, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float a = 0;
  register float b = 0;
  register float temp = 0;
 
  for (int j = 0; j < n; j++) {
    #pragma unroll 1
    for (int i = 0; i < k; i+=1){
      //load data
      a = *(A + lda * i);
      b = *(B + ldb * j + i);
      //compute
      temp += a * b;
      
    }
    *(C + j * ldc + idx) = temp;
    temp = 0;
  }
  
}


__global__ void
dgemm_kernel_reduce_gld(int m, int n, int k, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  //determine the row to process                                                        
  register int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float temp1 = 0;
  register float temp2 = 0;
  register float a = 0;
  register float b1 = 0;
  register float b2 = 0;

  #pragma unroll 1
  for (int i = 0; i < k; i+=1){
    //load data
    a = *A;
    b1 = *B;
    b2 = *(B + ldb);
    A += lda;
    B += 1;

    //compute
    temp1 = temp1 + a * b1;
    temp2 = temp2 + a * b2;

  }

  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  
}


void test_kernel_naive(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_naive<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V0");
}

void test_kernel_reduce_gld(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){
  
  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {
    // int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_reduce_gld<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V1");
}


/////////////////////////SHARED/////////////////////////
__global__ void
dgemm_kernel_shared(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ float cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  register float temp1 = 0;
  register float temp2 = 0;
  register float a = 0;

  for (int j = 0; j < k; j += T){
    cache[threadIdx.x * 2] = *(B + threadIdx.x);
    cache[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;
    for (int i = 0; i < T; i++) {
      a = *(A + (i + j) * lda);
      temp1 += a * cache[i * 2];
      temp2 += a * cache[i * 2 + 1];
    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;

}


float test_kernel_shared(int m, int n, int k, 
          float * dA, int lda, 
          float * dB, int ldb, 
          float * dC, int ldc,
          float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(1024, m); T *= 2) {

    //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel_shared<<<blocksPerGrid, threadsPerBlock,  T * sizeof(float) * 2>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V2");
}

///////////////////////A PREFETCH(cache<->register)
__global__ void
dgemm_kernel_prefetch_s2r_16(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{

  extern __shared__ float cache[];
  
  float * cacheA = cache;
  float * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  float r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);

      r8 = *(A + 8 *lda);
      r9 = *(A + 9 *lda);
      r10 = *(A + 10 *lda);
      r11 = *(A + 11 *lda);
      r12 = *(A + 12 *lda);
      r13 = *(A + 13 *lda);
      r14 = *(A + 14 *lda);
      r15 = *(A + 15 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 2];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 2 + 1];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;

      cacheA[threadIdx.x + 8 * T] = r8;
      cacheA[threadIdx.x + 9 * T] = r9;
      cacheA[threadIdx.x + 10 * T] = r10;
      cacheA[threadIdx.x + 11 * T] = r11;
      cacheA[threadIdx.x + 12 * T] = r12;
      cacheA[threadIdx.x + 13 * T] = r13;
      cacheA[threadIdx.x + 14 * T] = r14;
      cacheA[threadIdx.x + 15 * T] = r15;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;

}


__global__ void
dgemm_kernel_prefetch_s2r_8(int m, int n, int k, int T, float * A, int lda, float * B, int ldb, float * C, int ldc)
{

  extern __shared__ float cache[];
  
  float * cacheA = cache;
  float * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  float r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 2];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 2 + 1];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;

}


void test_kernel_prefetch(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 8; T <= 16; T *= 2) {
  //int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      if (T == 16)
        dgemm_kernel_prefetch_s2r_16<<<blocksPerGrid, threadsPerBlock, ((T * 2) + (T * T)) * sizeof(float)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      else if (T == 8)
        dgemm_kernel_prefetch_s2r_8<<<blocksPerGrid, threadsPerBlock, ((T * 2) + (T * T)) * sizeof(float)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-1");
}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, i, j, l (12)
//Double registers: cache, cacheA, cacheB, A, B, C, r0-3, temp1-2 (22)
//Shared mem.: T*2 + T*T (float)
__global__ void
dgemm_kernel_prefetch_s2r_4_16(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ float cache[];
 
  float * cacheA = cache;
  float * cacheB = cache + T * t; //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  float temp1 = 0;
  float temp2 = 0;

  #pragma unroll 1
  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;

  float r0, r1, r2, r3;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        r0 = *(A + 0 *lda);
        r1 = *(A + 1 *lda);
        r2 = *(A + 2 *lda);
        r3 = *(A + 3 *lda); 
      }

      #pragma unroll 1
      for (int i = 0; i < t; i++) {
        temp1 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i ];
        temp2 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 1];
      }
      if (l + t < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
    
}

void test_kernel_prefetch2(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++){
      dgemm_kernel_prefetch_s2r_4_16<<<blocksPerGrid, threadsPerBlock, ((T * 2) + (T * tt)) * sizeof(float)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    output(m, n, k, real_time, base, blocksPerGrid, threadsPerBlock, "V3-2");

}



//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (float)
//#define t 4
__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ float cacheB[];

  //determine the row to process                                                                                                                                                                                                                          
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  C = C + idx;
  register float temp1 = 0;
  register float temp2 = 0;

  register float nr0, nr1, nr2, nr3;
  register float cr0, cr1, cr2, cr3;

  //prefectch A 
  cr0 = *A;
  A += lda;
  cr1 = *A;
  A += lda;
  
  cr2 = *A;
  A += lda;
  cr3 = *A;
  A += lda;

  #pragma unroll 1
  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    // if (j == 0 && idx == 0)
    //   printf("%f %f \n", *(B + threadIdx.x), *(B + threadIdx.x + ldb));
    __syncthreads();
    B += T;

    #pragma unroll 1
    for (int l = j; l < j + T; l += 4){
      if (l + t < k) {
        nr0 = *A;
        A += lda;
        nr1 = *A;
        A += lda;

        nr2 = *A;
        A += lda;
        nr3 = *A;
        A += lda;
      }

      // if (j == 0 && idx == 0){
      //   printf("%f %f \n",  cacheB[(l - j) * 2 + 0 ], cacheB[(l - j) * 2 + 1]);
      //   printf("%f %f %f %f\n",  cr0, cr1, cr2 ,cr3);
      // }


      temp1 += cr0 * cacheB[(l - j) * 2 + 0 ];
      temp2 += cr0 * cacheB[(l - j) * 2 + 1];



      temp1 += cr1 * cacheB[(l - j) * 2 + 2 ];
      temp2 += cr1 * cacheB[(l - j) * 2 + 3];

     temp1 += cr2 * cacheB[(l - j) * 2 + 4 ];
     temp2 += cr2 * cacheB[(l - j) * 2 + 5];

     temp1 += cr3 * cacheB[(l - j) * 2 + 6];
     temp2 += cr3 * cacheB[(l - j) * 2 + 7];

   //   if (l + t < k) {
        cr0 = nr0;
        cr1 = nr1;
        cr2 = nr2;
        cr3 = nr3;
   //   }
    }
  }
  *C = temp1;
  *(C + ldc) = temp2;
    
}


//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, j, l (11)
//Double registers: cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (28)
//Shared mem.: T*2 + T*T (float)
__global__ void
dgemm_kernel4_3(int m, int n, int k, int T, int t, float * A, int lda, float * B, int ldb, float * C, int ldc)
{
                                                                                                                                                                                                                
  // //determine the row to process                                                                                                                                                                                                                          
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // A = A + idx;
  // C = C + idx;
  // register float temp1 = 0;
  // register float temp2 = 0;

  // register float nr0, nr1, nr2, nr3;
  // register float cr0, cr1, cr2, cr3;

  // register float nb00, nb01, nb10, nb11;
  // register float cb00, cb01, cb10, cb11;

  // //prefectch A 
  // cr0 = *A;
  // A += lda;
  // cr1 = *A;
  // A += lda;
  
  // cr2 = *A;
  // A += lda;
  // cr3 = *A;
  // A += lda;

  // cb00 = *B;
  // cb01 = *(B + ldb);
  // B += 1;
  // cb10 = *B;
  // cb11 = *(B + ldb);
  // B += 1;


  // #pragma unroll 1
  // for (int i = 0; i < k; i += t){ 
  //     if (i + t < k) {
  //       nr0 = *A;
  //       A += lda;
  //       nr1 = *A;
  //       A += lda;

  //       nr2 = *A;
  //       A += lda;
  //       nr3 = *A;
  //       A += lda;
  //     }
      
  //     //temp1 += cr1 * cr1;
  //     //temp2 += cr2 * cr2;
  //     //temp1 += cr3 * cr3;
  //     //temp1 += cr0 * cr0;
      
  //     nb00 = *B;
  //     nb01 = *(B + ldb);
  //     B += 1;
  //     nb10 = *B;
  //     nb11 = *(B + ldb);
  //     B += 1;

  //     temp1 += cr0 * cb00;
  //     temp2 += cr0 * cb01;
  //     temp1 += cr1 * cb10;
  //     temp2 += cr1 * cb11;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb10 = nb10;
  //     cb11 = nb11;


  //     if (i + t < k) {
  //       nb00 = *B;
  //       nb01 = *(B + ldb);
  //       B += 1;
  //       nb10 = *B;
  //       nb11 = *(B + ldb);
  //       B += 1;
  //     }

  //     temp1 += cr2 * cb00;
  //     temp2 += cr2 * cb01;
  //     temp1 += cr3 * cb10;
  //     temp2 += cr3 * cb11;

  //     cb00 = nb00;
  //     cb01 = nb01;
  //     cb10 = nb10;
  //     cb11 = nb11;
    
      
  //     if (i + t < k) {
  //       cr0 = nr0;
  //       cr1 = nr1;
  //       cr2 = nr2;
  //       cr3 = nr3;
  //     }
      
  // }
  // *C = temp1;
  // *(C + ldc) = temp2;
    
}


float test_kernel_prefetch3(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 4; T <= min(m, 1024); T*=2) {
 
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_2<<<blocksPerGrid, threadsPerBlock, ((T * 2)) * sizeof(float)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-3");
}


float test_kernel_prefetch4(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc,
            float base){

  float min_time = 1000;
  int blocksPerGrid_min, threadsPerBlock_min;
  for (int T = 16; T <= min(m, 1024); T*=2) {
  //int T = 128;
  int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4_3<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
      check_cuda_error();
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;
    if (real_time < min_time) {
      min_time = real_time; 
      blocksPerGrid_min = blocksPerGrid;
      threadsPerBlock_min = threadsPerBlock;
    }
  }
  output(m, n, k, min_time, base, blocksPerGrid_min, threadsPerBlock_min, "V3-4");
}




float test_cublas_mm(int m, int n, int k, 
            float * dA, int lda, 
            float * dB, int ldb, 
            float * dC, int ldc);


void test(int m, int k, int c);

int main(int argc, char *argv[]){
  for (int i = 10240; i <= 40960; i += 1024){
  //int i = 1024;
    // cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 2 << ")" << endl;
    test(i, i, atoi(argv[1]));
  }
}

void test(int m, int k, int c){
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 2;
    //int k = 20480;
    float * A = new float[m * k];
    float * B = new float[n * k];
    float * C = new float[m * n];
    float * checkC = new float[m * n];     

    for (int i = 0; i < m * k; i++){
    	A[i] = (float)i/(m * k);
    }

    for (int i = 0; i < n * k; i++){
    	B[i] = (float)i/(n * k);
    }
    
    float * dA;
    hipMalloc(&dA, m * k * sizeof(float));
    int lda = m;

    float * dB; 
    hipMalloc(&dB,  n * k * sizeof(float));
    int ldb = k;

    float * dC;
    hipMalloc(&dC, m * n * sizeof(float));
    int ldc = m;

    float * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(float));

    hipMemcpy(dA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(float), hipMemcpyHostToDevice);
    
    float base;

    base = test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
  
    if (c == -1) cout << base << endl;
    if (c == 0) test_kernel_naive(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 1) test_kernel_reduce_gld(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 2) test_kernel_shared(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch2(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    if (c == 3) test_kernel_prefetch3(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    // test_kernel_prefetch4(m, n, k, dA, lda, dB, ldb, dC, ldc, base);
    
   
    // cudaMemcpy(C, dC ,m * n * sizeof(float), cudaMemcpyDeviceToHost);
    // cudaMemcpy(checkC, dcheckC, m * n * sizeof(float), cudaMemcpyDeviceToHost);
    // //for (int i = 0; i < m * n; i++){
    // // cout<<C[i]<<" ";	
    // //}
    // check_C(C, m, n, checkC);

    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}



float test_cublas_mm(int m, int n, int k, 
         float * dA, int lda, 
         float * dB, int ldb, 
         float * dC, int ldc){

    float one = 1;
    float zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &one, dA, lda, dB, ldb, &zero, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;

    // cout <<"Runing time of culasdgemm:" << real_time <<" s." << endl;
    return real_time;
}






























